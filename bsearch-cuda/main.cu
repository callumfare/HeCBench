#include <cstdlib>
#include <chrono>
#include <iostream>
#include <hip/hip_runtime.h>

#ifndef Real_t 
#define Real_t float
#endif

//#define DEBUG // verify the results of kernel execution

template <typename T>
__global__ void
kernel_BS (const T* __restrict__ acc_a,
           const T* __restrict__ acc_z,
            size_t* __restrict__ acc_r,
           const size_t n)
{ 
  size_t i = blockIdx.x*blockDim.x+threadIdx.x;
  T z = acc_z[i];
  size_t low = 0;
  size_t high = n;
  while (high - low > 1) {
    size_t mid = low + (high - low)/2;
    if (z < acc_a[mid])
      high = mid;
    else
      low = mid;
  }
  acc_r[i] = low;
}

template <typename T>
__global__ void
kernel_BS2 (const T* __restrict__ acc_a,
            const T* __restrict__ acc_z,
             size_t* __restrict__ acc_r,
            const size_t n)
{
  size_t i = blockIdx.x*blockDim.x+threadIdx.x;
  unsigned  nbits = 0;
  while (n >> nbits) nbits++;
  size_t k = 1ULL << (nbits - 1);
  T z = acc_z[i];
  size_t idx = (acc_a[k] <= z) ? k : 0;
  while (k >>= 1) {
    size_t r = idx | k;
    if (r < n && z >= acc_a[r]) { 
      idx = r;
    }
  }
  acc_r[i] = idx;
}

template <typename T>
__global__ void
kernel_BS3 (const T* __restrict__ acc_a,
            const T* __restrict__ acc_z,
             size_t* __restrict__ acc_r,
            const size_t n)
{
  size_t i = blockIdx.x*blockDim.x+threadIdx.x;
  unsigned nbits = 0;
  while (n >> nbits) nbits++;
  size_t k = 1ULL << (nbits - 1);
  T z = acc_z[i];
  size_t idx = (acc_a[k] <= z) ? k : 0;
  while (k >>= 1) {
    size_t r = idx | k;
    size_t w = r < n ? r : n; 
    if (z >= acc_a[w]) { 
      idx = r;
    }
  }
  acc_r[i] = idx;
}

template <typename T>
__global__ void
kernel_BS4 (const T* __restrict__ acc_a,
            const T* __restrict__ acc_z,
             size_t* __restrict__ acc_r,
            const size_t n)
{
  __shared__  size_t k;

  size_t gid = blockIdx.x*blockDim.x+threadIdx.x;
  size_t lid = threadIdx.x; 

  if (lid == 0) {
    unsigned nbits = 0;
    while (n >> nbits) nbits++;
    k = 1ULL << (nbits - 1);
  }
  __syncthreads();

  size_t p = k;
  T z = acc_z[gid];
  size_t idx = (acc_a[p] <= z) ? p : 0;
  while (p >>= 1) {
    size_t r = idx | p;
    size_t w = r < n ? r : n;
    if (z >= acc_a[w]) { 
      idx = r;
    }
  }
  acc_r[gid] = idx;
}

template <typename T>
void bs ( const size_t aSize,
    const size_t zSize,
    const T *a,  // N+1
    const T *z,  // T
    size_t *r,   // T
    const size_t n )
{
  T* buf_x;
  T* buf_z;
  size_t *buf_r;
  hipMalloc((void**)&buf_x, sizeof(T)*aSize);
  hipMalloc((void**)&buf_z, sizeof(T)*zSize);
  hipMalloc((void**)&buf_r, sizeof(size_t)*zSize);
  hipMemcpy(buf_x, a,  sizeof(T)*aSize, hipMemcpyHostToDevice);
  hipMemcpy(buf_z, z,  sizeof(T)*zSize, hipMemcpyHostToDevice);
  kernel_BS<<<zSize/256, 256>>>(buf_x, buf_z, buf_r, n);
  hipMemcpy(r, buf_r, sizeof(size_t)*zSize, hipMemcpyDeviceToHost);
  hipFree(buf_x);
  hipFree(buf_z);
  hipFree(buf_r);
}

template <typename T>
void bs2 ( const size_t aSize,
    const size_t zSize,
    const T *a,  // N+1
    const T *z,  // T
    size_t *r,   // T
    const size_t n )
{
  T* buf_x;
  T* buf_z;
  size_t *buf_r;
  hipMalloc((void**)&buf_x, sizeof(T)*aSize);
  hipMalloc((void**)&buf_z, sizeof(T)*zSize);
  hipMalloc((void**)&buf_r, sizeof(size_t)*zSize);
  hipMemcpy(buf_x, a,  sizeof(T)*aSize, hipMemcpyHostToDevice);
  hipMemcpy(buf_z, z,  sizeof(T)*zSize, hipMemcpyHostToDevice);
  kernel_BS2<<<zSize/256, 256>>>(buf_x, buf_z, buf_r, n);
  hipMemcpy(r, buf_r, sizeof(size_t)*zSize, hipMemcpyDeviceToHost);
  hipFree(buf_x);
  hipFree(buf_z);
  hipFree(buf_r);
}

template <typename T>
void bs3 ( const size_t aSize,
    const size_t zSize,
    const T *a,  // N+1
    const T *z,  // T
    size_t *r,   // T
    const size_t n )
{
  T* buf_x;
  T* buf_z;
  size_t *buf_r;
  hipMalloc((void**)&buf_x, sizeof(T)*aSize);
  hipMalloc((void**)&buf_z, sizeof(T)*zSize);
  hipMalloc((void**)&buf_r, sizeof(size_t)*zSize);
  hipMemcpy(buf_x, a,  sizeof(T)*aSize, hipMemcpyHostToDevice);
  hipMemcpy(buf_z, z,  sizeof(T)*zSize, hipMemcpyHostToDevice);
  kernel_BS3<<<zSize/256, 256>>>(buf_x, buf_z, buf_r, n);
  hipMemcpy(r, buf_r, sizeof(size_t)*zSize, hipMemcpyDeviceToHost);
  hipFree(buf_x);
  hipFree(buf_z);
  hipFree(buf_r);
}

template <typename T>
void bs4 ( const size_t aSize,
    const size_t zSize,
    const T *a,  // N+1
    const T *z,  // T
    size_t *r,   // T
    const size_t n )
{
  T* buf_x;
  T* buf_z;
  size_t *buf_r;
  hipMalloc((void**)&buf_x, sizeof(T)*aSize);
  hipMalloc((void**)&buf_z, sizeof(T)*zSize);
  hipMalloc((void**)&buf_r, sizeof(size_t)*zSize);
  hipMemcpy(buf_x, a,  sizeof(T)*aSize, hipMemcpyHostToDevice);
  hipMemcpy(buf_z, z,  sizeof(T)*zSize, hipMemcpyHostToDevice);
  kernel_BS4<<<zSize/256, 256>>>(buf_x, buf_z, buf_r, n);
  hipMemcpy(r, buf_r, sizeof(size_t)*zSize, hipMemcpyDeviceToHost);
  hipFree(buf_x);
  hipFree(buf_z);
  hipFree(buf_r);
}

#ifdef DEBUG
void verify(Real_t *a, Real_t *z, size_t *r, size_t aSize, size_t zSize, std::string msg)
{
  for (size_t i = 0; i < zSize; ++i)
  {
    // check result
    if (!(r[i]+1 < aSize && a[r[i]] <= z[i] && z[i] < a[r[i] + 1]))
    {
      std::cout << msg << ": incorrect result:" << std::endl;
      std::cout << "index = " << i << " r[index] = " << r[i] << std::endl;
      std::cout << a[r[i]] << " <= " << z[i] << " < " << a[r[i] + 1] << std::endl;
      break;
    }
    // clear result
    r[i] = 0xFFFFFFFF;
  }
}
#endif

int main(int argc, char* argv[])
{
  if (argc != 3) {
    std::cout << "Usage ./main <number of elements> <repeat>\n";
    return 1;
  }

  size_t numElem = atol(argv[1]);
  uint repeat = atoi(argv[2]);

  srand(2);
  size_t aSize = numElem;
  size_t zSize = 2*aSize;
  Real_t *a = NULL;
  Real_t *z = NULL;
  size_t *r = NULL;
  posix_memalign((void**)&a, 1024, aSize * sizeof(Real_t));
  posix_memalign((void**)&z, 1024, zSize * sizeof(Real_t));
  posix_memalign((void**)&r, 1024, zSize * sizeof(size_t));

  size_t N = aSize-1;

  // strictly ascending
  for (size_t i = 0; i < aSize; i++) a[i] = i;

  // lower = 0, upper = n-1
  for (size_t i = 0; i < zSize; i++) { 
    z[i] = rand() % N;
  }

  auto start = std::chrono::steady_clock::now();
  for(uint k = 0; k < repeat; k++) {
    bs(aSize, zSize, a, z, r, N);  
  }
  auto end = std::chrono::steady_clock::now();
  auto time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  std::cout << "Average device execution time (bs1) " << (time * 1e-9f) / repeat << " (s)\n";

#ifdef DEBUG
  verify(a, z, r, aSize, zSize, "bs1");
#endif

  start = std::chrono::steady_clock::now();
  for(uint k = 0; k < repeat; k++) {
    bs2(aSize, zSize, a, z, r, N);  
  }
  end = std::chrono::steady_clock::now();
  time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  std::cout << "Average device execution time (bs2) " << (time * 1e-9f) / repeat << " (s)\n";

#ifdef DEBUG
  verify(a, z, r, aSize, zSize, "bs2");
#endif

  start = std::chrono::steady_clock::now();
  for(uint k = 0; k < repeat; k++) {
    bs3(aSize, zSize, a, z, r, N);  
  }
  end = std::chrono::steady_clock::now();
  time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  std::cout << "Average device execution time (bs3) " << (time * 1e-9f) / repeat << " (s)\n";

#ifdef DEBUG
  verify(a, z, r, aSize, zSize, "bs3");
#endif

  start = std::chrono::steady_clock::now();
  for(uint k = 0; k < repeat; k++) {
    bs4(aSize, zSize, a, z, r, N);  
  }
  end = std::chrono::steady_clock::now();
  time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  std::cout << "Average device execution time (bs4) " << (time * 1e-9f) / repeat << " (s)\n";

#ifdef DEBUG
  verify(a, z, r, aSize, zSize, "bs4");
#endif

  free(a);
  free(z);
  free(r);
  return 0;
}
